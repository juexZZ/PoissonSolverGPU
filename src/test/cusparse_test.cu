#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""
#include "hipsparse.h"
#include "hipblas.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <Eigen/Dense>
#include <Eigen/Sparse>
#include <unsupported/Eigen/SparseExtra>
#include "../utils.h"
#include "../solver/cusparsesolver.cuh"
using namespace std;
using namespace Eigen;

int main(int argc, char* argv[]) {
	if (argc != 2) {
		cout << "Usage: ./cudatest num_iter" << endl;
	}

	/*string PATH = "../../data/";*/
	string PATH = "C:/NYU/gpu/PoissonSolver/data/";
	Eigen::SparseMatrix<double> A;
	readSymetric(A, PATH + "nos6.mtx");
	A.makeCompressed();
	VectorXd rhs(A.cols()), x(A.rows()), x_cg(A.rows());
	rhs.setOnes();
	x.setZero();
	double reTol = 1e-8; //Relative error tolerence
	int maxIter = atoi(argv[1]);
	hipError_t cudaStatus;
	unsigned int N = A.rows();
	unsigned int vector_bytesize = N * sizeof(double);
	double abstol = reTol * reTol * rhs.norm();
	// convert matrix to row-major storage
	// allocate and move to device
	double* rhs_d; // b(rhs) on device
	// Triplet for A
	double* A_d; //csrValA
	int* ia_d; //csrRowPtrA
	int* ja_d;//csrColIndA 
	double* r_k_norm;
	double* r_k1_norm;
	double* pAp_k;

	double* x_d; //x on device
	double* Ap_rd;
	double* Ap_r;
	double initial = 0.0;
	hipsparseHandle_t handle;
	hipsparseSpMatDescr_t* A_sparse;
	hipsparseCreateCsr(A_sparse,
		N,
		N,
		A.nonZeros(),
		ia_d,
		ja_d,
		A_d,
		HIPSPARSE_INDEX_32I,
		HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);

	cudaStatus = hipMalloc((void**)&A_d, A.nonZeros() * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		//goto Error;
	}
	//csrRowPtrA
	cudaStatus = hipMalloc((void**)&ia_d, (A.rows() + 1) * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		//goto Error;
	}
	cudaStatus = hipMalloc((void**)&ja_d, A.nonZeros() * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		//goto Error;
	}

	cudaStatus = hipMalloc((void**)&rhs_d, vector_bytesize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		//goto Error;
	}
	cudaStatus = hipMalloc((void**)&x_d, vector_bytesize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		//goto Error;
	}
	cudaStatus = hipMalloc((void**)&Ap_rd, vector_bytesize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		//goto Error;
	}

	cudaStatus = hipMalloc((void**)&r_k_norm, sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		//goto Error;
	}
	cudaStatus = hipMalloc((void**)&r_k1_norm, sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		//goto Error;
	}
	cudaStatus = hipMalloc((void**)&pAp_k, sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		//goto Error;
	}

	/*cudaStatus = hipMemcpy(a_d, a.begin_, A.nonZeros() * sizeof(double), hipMemcpyHostToDevice);*/
	cudaStatus = hipMemcpy(A_d, A.valuePtr(), A.nonZeros() * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		//goto Error;
	}
	/*cudaStatus = hipMemcpy(ia_d, ia.begin_, (A.rows() + 1) * sizeof(int), hipMemcpyHostToDevice);*/
	cudaStatus = hipMemcpy(ia_d, A.outerIndexPtr(), (A.rows() + 1) * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		//goto Error;
	}
	/*cudaStatus = hipMemcpy(ja_d, ja.begin_, A.nonZeros() * sizeof(int), hipMemcpyHostToDevice);*/
	cudaStatus = hipMemcpy(ja_d, A.innerIndexPtr(), A.nonZeros() * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		//goto Error;
	}


	cudaStatus = hipMemcpy(rhs_d, rhs.data(), vector_bytesize, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		//goto Error;
	}
	cudaStatus = hipMemcpy(x_d, x.data(), vector_bytesize, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		//goto Error;
	}
	cudaStatus = hipMemcpy(Ap_rd, Ap_r, vector_bytesize, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		//goto Error;
	}
	cudaStatus = hipMemcpy(r_k_norm, &initial, sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		//goto Error;
	}
	cudaStatus = hipMemcpy(r_k1_norm, &initial, sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		//goto Error;
	}
	cudaStatus = hipMemcpy(pAp_k, &initial, sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		//goto Error;
	}

	// create intermediate variables
	double* rk; //residue
	double* pk;
	cudaStatus = hipMalloc((void**)&rk, vector_bytesize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		//goto Error;
	}
	cudaStatus = hipMalloc((void**)&pk, vector_bytesize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		//goto Error;
	}

	//setup geometry
	unsigned int threadsPerBlock = 256;
	unsigned int blocksPerGrid = ceil((double)N / 256.0);

	// solve at device side
	wrapper_PoissonSolverSparse_cusparese(blocksPerGrid, threadsPerBlock,A_sparse,rhs_d, A_d, ia_d, ja_d, x_d, rk, pk, abstol, N, maxIter, Ap_rd, r_k_norm, r_k1_norm, pAp_k);

	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		printf("CUDA error: %s\n", hipGetErrorString(error));
	}
	else {
		printf("CUDA NO error\n");
	}
	// move back and write to the root vector
	VectorXd root(N);
	hipMemcpy(root.data(), x_d, vector_bytesize, hipMemcpyDeviceToHost);
	//free and error handle

	hipFree(A_d);
	hipFree(ia_d);
	hipFree(ja_d);
	hipFree(rhs_d);
	hipFree(x_d);
	hipFree(rk);
	hipFree(pk);

	// check
	double err = (A * root - rhs).norm();
	if (err < 1e-5) {
		cout << "pass with err:" << err << endl;
	}
	else {
		cout << "not pass with err:" << err << endl;
		cout << "current solution: \n" << root << endl;
	}