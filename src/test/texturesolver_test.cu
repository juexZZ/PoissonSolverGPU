#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <Eigen/Dense>
#include <Eigen/Sparse>
#include <unsupported/Eigen/SparseExtra>
#include <iostream>
#include <string>
#include "../utils.h"
#include "../solver/texturesolver.cuh"

using namespace std;



int main(int argc, char* argv[]) {
	if(argc!=2){
		cout<<"Usage: ./cudatest num_iter"<<endl;
	}
	/*string PATH = "../../data/";*/
	string PATH = "C:/NYU/gpu/PoissonSolver/data/";
	Eigen::MatrixXf A = openData<float>(PATH + "test_100.csv");
	Eigen::VectorXf rhs(A.cols()), x(A.rows()), x_cg(A.rows());
	rhs.setOnes();
	x.setZero();
	//for (size_t i = 0; i < A.rows(); i++)
	//{
	//	A.row(i) = A.row(i) / A(i,i);
	//	rhs[i] = rhs[i] / A(i,i);
	//}
	float reTol = 1e-8; //Relative error tolerence
	//float absTol = 0;
	int maxIter = atoi(argv[1]);
	hipError_t cudaStatus;
	unsigned int matrix_bytesize = A.size() * sizeof(float); // NxN
	unsigned int N = A.rows();
	unsigned int vector_bytesize = N * sizeof(float);
	float abstol = reTol * reTol * rhs.norm();
	// convert matrix to row-major storage
	Eigen::Matrix<float, Eigen::Dynamic, Eigen::Dynamic, Eigen::RowMajor> Arowmajor = A;
	// allocate and move to device
	float* rhs_d; // b(rhs) on device
	float* A_d; // A on device
	float* x_d; //x on device
	cudaStatus = hipMalloc((void**)&A_d, matrix_bytesize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		//goto Error;
	}
	cudaStatus = hipMalloc((void**)&rhs_d, vector_bytesize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		//goto Error;
	}
	cudaStatus = hipMalloc((void**)&x_d, vector_bytesize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		//goto Error;
	}

	cudaStatus = hipMemcpy(A_d, Arowmajor.data(), matrix_bytesize, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		//goto Error;
	}
	cudaStatus = hipMemcpy(rhs_d, rhs.data(), vector_bytesize, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		//goto Error;
	}
	cudaStatus = hipMemcpy(x_d, x.data(), vector_bytesize, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		//goto Error;
	}

	// create intermediate variables
	float* rk; //residue
	float* pk;
	cudaStatus = hipMalloc((void**)&rk, vector_bytesize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		//goto Error;
	}
	cudaStatus = hipMalloc((void**)&pk, vector_bytesize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		//goto Error;
	}

	//setup geometry
	unsigned int threadsPerBlock = N;
	unsigned int blocksPerGrid = 1;

	// solve at device side
	wrapper_PoissonSolverTexture(blocksPerGrid, threadsPerBlock, rhs_d, A_d, x_d, rk, pk, abstol, N, maxIter);

	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if(error!=hipSuccess){
		printf("CUDA error: %s\n", hipGetErrorString(error));
    }else{
		printf("CUDA NO error\n");
	}
	// move back and write to the root vector
	Eigen::VectorXf root(N);
	hipMemcpy(root.data(), x_d, vector_bytesize, hipMemcpyDeviceToHost);
	//free and error handle

	hipFree(A_d);
	hipFree(rhs_d);
	hipFree(x_d);
	hipFree(rk);
	hipFree(pk);

	// check
	float err = (A * root - rhs).norm();
	if (err < 1e-5){
		cout << "pass with err:"<< err << endl;
	}
	else {
		cout << "not pass with err:"<< err << endl;
		cout << "current solution: \n"<<root<< endl;
	}
}